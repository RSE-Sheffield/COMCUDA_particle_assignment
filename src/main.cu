#include <stdlib.h>
#include <ctype.h>
#include <string.h>
#include <hip/hip_runtime.h>

#ifdef _MSC_VER
#include <windows.h>
#include <WinCon.h>
#endif

#include <random>

#define CONSOLE_RED "\x1b[91m"
#define CONSOLE_GREEN "\x1b[92m"
#define CONSOLE_YELLOW "\x1b[93m"
#define CONSOLE_RESET "\x1b[39m"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "external/stb_image_write.h"

#include "main.h"
#include "config.h"
#include "common.h"
#include "cpu.h"
#include "openmp.h"
#include "cuda.cuh"
#include "helper.h"

int main(int argc, char **argv)
{
#ifdef _MSC_VER
    {
        HANDLE hConsole = GetStdHandle(STD_OUTPUT_HANDLE);
        DWORD consoleMode;
        GetConsoleMode(hConsole, &consoleMode);
        consoleMode |= ENABLE_VIRTUAL_TERMINAL_PROCESSING;  // Enable support for ANSI colours (Windows 10+)
        SetConsoleMode(hConsole, consoleMode);
    }
#endif
    // Parse args
    Config config;
    parse_args(argc, argv, &config);

    // Load input config
    InputFile user_config;
    {
        //user_cimage.data = stbi_load(config.input_file, &user_cimage.width, &user_cimage.height, &user_cimage.channels, 0);
        //if (!user_cimage.data) {
        //    printf("Unable to load image '%s', please try a different file.\n", config.input_file);
        //    return EXIT_FAILURE;
        //}
        //if (user_cimage.channels == 2) {
        //    printf("2 channel images are not supported, please try a different file.\n");
        //    return EXIT_FAILURE;
        //}
    }

    // Generate Initial Particles from user_config
    const unsigned int particles_count = user_config.circle_count + user_config.square_count;
    Particle* particles = (Particle *)malloc(particles_count * sizeof(Particle));
    {
        // Random engine with a fixed seed and several distributions to be used
        std::mt19937 rng(12);
        std::uniform_real_distribution<float> normalised_float_dist(0, 1);
        std::normal_distribution<float> circle_rad_dist(user_config.circle_rad_average, user_config.circle_rad_standarddev);
        std::normal_distribution<float> circle_opacity_dist(user_config.circle_rad_average, user_config.circle_rad_standarddev);
        std::normal_distribution<float> square_rad_dist(user_config.square_rad_average, user_config.square_rad_standarddev);
        std::normal_distribution<float> square_opacity_dist(user_config.square_rad_average, user_config.square_rad_standarddev);
        std::uniform_int_distribution<int> color_palette_dist(0, sizeof(base_color_palette)/sizeof(unsigned char[3]) - 1);
        // Common
        for (unsigned int i = user_config.circle_count; i < particles_count; ++i) {
            const int palette_index = color_palette_dist(rng);
            particles[i].color[0] = base_color_palette[palette_index][0];
            particles[i].color[1] = base_color_palette[palette_index][1];
            particles[i].color[2] = base_color_palette[palette_index][2];
            particles[i].location[0] = normalised_float_dist(rng) * OUT_IMAGE_WIDTH;
            particles[i].location[1] = normalised_float_dist(rng) * OUT_IMAGE_HEIGHT;
            particles[i].location[2] = normalised_float_dist(rng);
            // particles[i].direction[0]
            // particles[i].direction[1]
            // particles[i].speed
        }
        // Circles
        for (unsigned int i = 0; i < user_config.circle_count; ++i) {
            particles[i].type = Circle;
            particles[i].radius = circle_rad_dist(rng);
            particles[i].opacity = circle_opacity_dist(rng);
        }
        // Squares
        for (unsigned int i = user_config.circle_count; i < particles_count; ++i) {
            particles[i].type = Square;
            particles[i].radius = square_rad_dist(rng);
            particles[i].opacity = square_opacity_dist(rng);
        }
        // Clamp radius/opacity to bounds (use OpenMP in an attempt to trigger OpenMPs hidden init cost)
#pragma omp parallel for 
        for (unsigned int i = user_config.circle_count; i < particles_count; ++i) {
            particles[i].radius = particles[i].radius < MIN_RADIUS ? MIN_RADIUS : particles[i].radius;
            particles[i].radius = particles[i].radius > MAX_RADIUS ? MAX_RADIUS : particles[i].radius;
            particles[i].opacity = particles[i].opacity < MIN_OPACITY ? MIN_OPACITY : particles[i].opacity;
            particles[i].opacity = particles[i].opacity > MAX_OPACITY ? MAX_OPACITY : particles[i].opacity;
        }
    }

    // Create result for validation
    CImage validation_image;
    {
        // @TODO
    }
       
    CImage output_image;
    Runtimes timing_log;
    const int TOTAL_RUNS = config.benchmark ? BENCHMARK_RUNS : 1;
    {
        //Init for run  
        hipEvent_t startT, initT, stage1T, stage2T, stage3T, stopT;
        CUDA_CALL(hipEventCreate(&startT));
        CUDA_CALL(hipEventCreate(&initT));
        CUDA_CALL(hipEventCreate(&stage1T));
        CUDA_CALL(hipEventCreate(&stage2T));
        CUDA_CALL(hipEventCreate(&stage3T));
        CUDA_CALL(hipEventCreate(&stopT));

        // Run 1 or many times
        memset(&timing_log, 0, sizeof(Runtimes));
        for (int runs = 0; runs < TOTAL_RUNS; ++runs) {
            if (TOTAL_RUNS > 1)
                printf("\r%d/%d", runs + 1, TOTAL_RUNS);
            memset(&output_image, 0, sizeof(CImage));
            output_image.data = (unsigned char*)malloc(OUT_IMAGE_WIDTH * OUT_IMAGE_HEIGHT * sizeof(unsigned char));
            memset(output_image.data, 0, OUT_IMAGE_WIDTH * OUT_IMAGE_HEIGHT * sizeof(unsigned char));
            // Run Adaptive Histogram algorithm
            CUDA_CALL(hipEventRecord(startT));
            CUDA_CALL(hipEventSynchronize(startT));
            switch (config.mode) {
            case CPU:
                {
                    cpu_begin(particles, particles_count);
                    CUDA_CALL(hipEventRecord(initT));
                    CUDA_CALL(hipEventSynchronize(initT));
                    cpu_stage1();
                    CUDA_CALL(hipEventRecord(stage1T));
                    CUDA_CALL(hipEventSynchronize(stage1T));
                    cpu_stage2();
                    CUDA_CALL(hipEventRecord(stage2T));
                    CUDA_CALL(hipEventSynchronize(stage2T));
                    cpu_stage3();
                    CUDA_CALL(hipEventRecord(stage3T));
                    CUDA_CALL(hipEventSynchronize(stage3T));
                    cpu_end(&output_image);
                }
                break;
            case OPENMP:
                {
                    openmp_begin(particles, particles_count);
                    CUDA_CALL(hipEventRecord(initT));
                    CUDA_CALL(hipEventSynchronize(initT));
                    openmp_stage1();
                    CUDA_CALL(hipEventRecord(stage1T));
                    CUDA_CALL(hipEventSynchronize(stage1T));
                    openmp_stage2();
                    CUDA_CALL(hipEventRecord(stage2T));
                    CUDA_CALL(hipEventSynchronize(stage2T));
                    openmp_stage3();
                    CUDA_CALL(hipEventRecord(stage3T));
                    CUDA_CALL(hipEventSynchronize(stage3T));
                    openmp_end(&output_image);
                }
                break;
            case CUDA:
                {
                    cuda_begin(particles, particles_count);
                    CUDA_CHECK();
                    CUDA_CALL(hipEventRecord(initT));
                    CUDA_CALL(hipEventSynchronize(initT));
                    cuda_stage1();
                    CUDA_CHECK();
                    CUDA_CALL(hipEventRecord(stage1T));
                    CUDA_CALL(hipEventSynchronize(stage1T));
                    cuda_stage2();
                    CUDA_CHECK();
                    CUDA_CALL(hipEventRecord(stage2T));
                    CUDA_CALL(hipEventSynchronize(stage2T));
                    cuda_stage3();
                    CUDA_CHECK();
                    CUDA_CALL(hipEventRecord(stage3T));
                    CUDA_CALL(hipEventSynchronize(stage3T));
                    cuda_end(&output_image);
                }
                break;
            }
            CUDA_CALL(hipEventRecord(stopT));
            CUDA_CALL(hipEventSynchronize(stopT));
            // Sum timing info
            float milliseconds = 0;
            CUDA_CALL(hipEventElapsedTime(&milliseconds, startT, initT));
            timing_log.init += milliseconds;
            CUDA_CALL(hipEventElapsedTime(&milliseconds, initT, stage1T));
            timing_log.stage1 += milliseconds;
            CUDA_CALL(hipEventElapsedTime(&milliseconds, stage1T, stage2T));
            timing_log.stage2 += milliseconds;
            CUDA_CALL(hipEventElapsedTime(&milliseconds, stage2T, stage3T));
            timing_log.stage3 += milliseconds;
            CUDA_CALL(hipEventElapsedTime(&milliseconds, stage3T, stopT));
            timing_log.cleanup += milliseconds;
            CUDA_CALL(hipEventElapsedTime(&milliseconds, startT, stopT));
            timing_log.total += milliseconds;
            // Avoid memory leak
            if (runs + 1 < TOTAL_RUNS) {
                if (output_image.data)
                    free(output_image.data);
            }
        }
        // Convert timing info to average
        timing_log.init /= TOTAL_RUNS;
        timing_log.stage1 /= TOTAL_RUNS;
        timing_log.stage2 /= TOTAL_RUNS;
        timing_log.stage3 /= TOTAL_RUNS;
        timing_log.cleanup /= TOTAL_RUNS;
        timing_log.total /= TOTAL_RUNS;

        // Cleanup timing
        hipEventDestroy(startT);
        hipEventDestroy(initT);
        hipEventDestroy(stage1T);
        hipEventDestroy(stage2T);
        hipEventDestroy(stage3T);
        hipEventDestroy(stopT);
    }

    // Validate and report    
    {
        printf("\rValidation Status: \n");
        printf("\tImage width: %s" CONSOLE_RESET "\n", validation_image.width == output_image.width ? CONSOLE_GREEN "Pass" : CONSOLE_RED "Fail");
        printf("\tImage height: %s" CONSOLE_RESET "\n", validation_image.height == output_image.height ? CONSOLE_GREEN "Pass" : CONSOLE_RED "Fail");
        int v_size = validation_image.width * validation_image.height;
        int o_size = output_image.width * output_image.height;
        int s_size = v_size < o_size ? v_size : o_size;
        int bad_pixels = 0;
        int close_pixels = 0;
        if (output_image.data && s_size) {
            for (int i = 0; i < s_size; ++i) {
                if (output_image.data[i] != validation_image.data[i]) {
                    // Give a +-1 threshold for error (incase fast-math triggers a small difference in places)
                    if (output_image.data[i]+1 == validation_image.data[i] || output_image.data[i]-1 == validation_image.data[i]) {
                        close_pixels++;
                    } else {
                        bad_pixels++;
                    }
                }
            }
            printf("\tImage pixels: ");
            if (bad_pixels) {
                printf(CONSOLE_RED "Fail" CONSOLE_RESET " (%d/%u wrong)\n", bad_pixels, o_size);
            } else {
                printf(CONSOLE_GREEN "Pass" CONSOLE_RESET "\n");
            }
        } else {
            printf("\tImage pixels: " CONSOLE_RED "Fail" CONSOLE_RESET "\n");
        }
    }

    // Export output image
    if (config.output_file) {
        if (!stbi_write_png(config.output_file, output_image.width, output_image.height, output_image.channels, output_image.data, output_image.width * output_image.channels)) {
            printf(CONSOLE_YELLOW "Unable to save image output to %s.\n" CONSOLE_RESET, config.output_file);
            // return EXIT_FAILURE;
        }
    }


    // Report timing information    
    printf("%s Average execution timing from %d runs\n", mode_to_string(config.mode), TOTAL_RUNS);
    if (config.mode == CUDA) {
        int device_id = 0;
        CUDA_CALL(hipGetDevice(&device_id));
        hipDeviceProp_t props;
        memset(&props, 0, sizeof(hipDeviceProp_t));
        CUDA_CALL(hipGetDeviceProperties(&props, device_id));
        printf("Using GPU: %s\n", props.name);
    }
#ifdef _DEBUG
    printf(CONSOLE_YELLOW "Code built as DEBUG, timing results are invalid!\n" CONSOLE_RESET);
#endif
    printf("Init: %.3fms\n", timing_log.init);
    printf("Stage 1: %.3fms%s\n", timing_log.stage1, getStage1SkipUsed() ? CONSOLE_YELLOW " (helper method used, time invalid)" CONSOLE_RESET : "");
    printf("Stage 2: %.3fms%s\n", timing_log.stage2, getStage2SkipUsed() ? CONSOLE_YELLOW " (helper method used, time invalid)" CONSOLE_RESET : "");
    printf("Stage 3: %.3fms%s\n", timing_log.stage3, getStage3SkipUsed() ? CONSOLE_YELLOW " (helper method used, time invalid)" CONSOLE_RESET : "");
    printf("Free: %.3fms\n", timing_log.cleanup);
    printf("Total: %.3fms%s\n", timing_log.total, getSkipUsed() ? CONSOLE_YELLOW " (helper method used, time invalid)" CONSOLE_RESET : "");

    // Cleanup
    hipDeviceReset();
    free(validation_image.data);
    free(particles);
    free(output_image.data);
    free(config.input_file);
    if (config.output_file)
        free(config.output_file);
    return EXIT_SUCCESS;
}
void parse_args(int argc, char **argv, Config *config) {
    // Clear config struct
    memset(config, 0, sizeof(Config));
    if (argc < 3 || argc > 5) {
        fprintf(stderr, "Program expects 2-4 arguments, only %d provided.\n", argc-1);
        print_help(argv[0]);
    }
    // Parse first arg as mode
    {
        char lower_arg[7];  // We only care about first 6 characters
        // Convert to lower case
        int i = 0;
        for(; argv[1][i] && i < 6; i++){
            lower_arg[i] = tolower(argv[1][i]);
        }
        lower_arg[i] = '\0';
        // Check for a match
        if (!strcmp(lower_arg, "cpu")) {
            config->mode = CPU;
        } else if (!strcmp(lower_arg, "openmp")) {
            config->mode = OPENMP;
        } else if (!strcmp(lower_arg, "cuda") || !strcmp(lower_arg, "gpu")) {
            config->mode = CUDA;
        } else {
            fprintf(stderr, "Unexpected string provided as first argument: '%s' .\n", argv[1]);
            fprintf(stderr, "First argument expects a single mode as string: CPU, OPENMP, CUDA.\n");
            print_help(argv[0]);
        }
    }
    // Parse second arg as input file
    {
        // Find length of string
        const size_t input_name_len = strlen(argv[2]) + 1;  // Add 1 for null terminating character
        // Allocate memory and copy
        config->input_file = (char*)malloc(input_name_len);
        memcpy(config->input_file, argv[2], input_name_len);
    }
    
    // Iterate over remaining args    
    int i = 3;
    char * t_arg = 0;
    for (; i < argc; i++) {
        // Make a lowercase copy of the argument
        const size_t arg_len = strlen(argv[i]) + 1;  // Add 1 for null terminating character
        if (t_arg) 
            free(t_arg);
        t_arg = (char*)malloc(arg_len);
        int j = 0;
        for(; argv[i][j]; ++j){
            t_arg[j] = tolower(argv[i][j]);
        }
        t_arg[j] = '\0';
        // Decide which arg it is
        if (!strcmp("--bench", t_arg) || !strcmp("--benchmark", t_arg)|| !strcmp("-b", t_arg)) {
            config->benchmark = 1;
            continue;
        }
        if (!strcmp(t_arg + arg_len - 5, ".png")) {
            // Allocate memory and copy
            config->output_file = (char*)malloc(arg_len);
            memcpy(config->output_file, argv[i], arg_len);
            continue;
        }
        fprintf(stderr, "Unexpected optional argument: %s\n", argv[i]);
        print_help(argv[0]);
    }
    if (t_arg) 
        free(t_arg);
}
void print_help(const char *program_name) {
    fprintf(stderr, "%s <mode> <input image> (<output image>) (--bench)\n", program_name);
    
    const char *line_fmt = "%-18s %s\n";
    fprintf(stderr, "Required Arguments:\n");
    fprintf(stderr, line_fmt, "<mode>", "The algorithm to use: CPU, OPENMP, CUDA");
    fprintf(stderr, line_fmt, "<input image>", "Input image, .png, .jpg");
    fprintf(stderr, "Optional Arguments:\n");
    fprintf(stderr, line_fmt, "<output image>", "Output image, requires .png filetype");
    fprintf(stderr, line_fmt, "-b, --bench", "Enable benchmark mode");

    exit(EXIT_FAILURE);
}
const char *mode_to_string(Mode m) {
    switch (m)
    {
    case CPU:
      return "CPU";
    case OPENMP:
     return "OpenMP";
    case CUDA:
      return "CUDA";
    }
    return "?";
}
